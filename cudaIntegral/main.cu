// #define _GNU_SOURCE             /* See feature_test_macros(7) */

#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <pthread.h>
#include <sched.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <string.h>
#include <iostream>
#include <hip/hip_runtime.h>




#define ALIGN 64
#define MAXCPUS 20
#define COREID "processor	:"
#define PHYSID "core id		:"

#define f(x) (x*x*x +2*x*x-8*x-1)
/*
#define X_START (-1)
#define X_END (1)
#define CUTS 100
*/
#define _(var) do {errno = 0; \
			if ((var)<0) \
				{ perror(#var); return -1;} }while(0)

#define PL fprintf(stderr, "%d\n", __LINE__);

#define min(a, b) (a < b ? a : b)
#define max(a, b) (a > b ? a : b)
typedef struct CalcArgs
{
	double sum;
	unsigned long long cutNumber;
	double xStart;
	double xEnd;
}CalcArgs;



typedef struct CpuIds
{
	int phys;
	int core;
}CpuIds;

double X_START = -2;
double X_END = 2;
unsigned long long CUTS = (unsigned long long)360360000ull;//*10;
int cpuTable[MAXCPUS][MAXCPUS] = {};
int cpuPhysCoreMax[MAXCPUS] = {};
int cpuPhysCoreMin[MAXCPUS] = {};

int cpuCorePhys[MAXCPUS] = {};
CpuIds Idtable[MAXCPUS] = {};
int maxPhys = 0;

int CpuTableDump()
{
	for (int i = 0; i < MAXCPUS; i++)
	{
		printf("%3d:\t", i);

		for (int j = 0; j < MAXCPUS; j++)
			printf("%2d ", cpuTable[i][j]);
		printf("\n");
	}
	return 0;
}

__global__
void calculate(CalcArgs* args)
{
	CalcArgs* arg = args /*+ blockIdx.x * blockDim.x*/ + threadIdx.x;
	// fprintf(stderr, "%p\n", &arg->tid);
	double sum = 0;
	double xEnd =  arg->xEnd;
	double xStart = arg->xStart;
	double dx = ( xEnd - xStart ) / arg->cutNumber;
	for (double i = xStart; i <= xEnd; i += dx)
		sum += f(i) * dx;
	arg->sum = sum;

	return;
}

void* bisy(void* args)
{
	while(1);
}


int main(int argc, char const *argv[])
{
	if (argc != 2)
	{
		printf("Usage: %s <number of threads(>=1)>\n", argv[0]);
		return EXIT_FAILURE;
	}
	long nThreads = strtol(argv[1], NULL, 10);
	if (nThreads < 1)
	{
		printf("Usage: %s <number of threads(>=1)>\n", argv[0]);
		return EXIT_FAILURE;
	}


	CalcArgs* args = (CalcArgs*) malloc(nThreads * sizeof(CalcArgs));

	//fprintf(stderr, "%p %d %lu, %ld\n", args, err, sizeof(CalcArgs), nThreads);

	int maxThreads = sysconf(_SC_NPROCESSORS_ONLN);

	unsigned long long eachCutNum = CUTS / min(nThreads, maxThreads);
	double cutLong = (X_END - X_START)/ nThreads;
	for (int i = 0; i < nThreads; i++)
	{

		args[i].xStart = X_START + cutLong*i;
		args[i].xEnd = args[i].xStart + cutLong;
		args[i].cutNumber = eachCutNum ;// (nThreads/maxThreads + (i%maxThreads<nThreads%maxThreads? 1 : 0) );
		args[i].sum = 0;
		

	}
	double sum = 0;
	hipSetDevice(0);

	CalcArgs* argsH = NULL;
	hipMalloc(&argsH, nThreads * sizeof(CalcArgs));
	hipMemcpy(argsH, args, nThreads * sizeof(CalcArgs), hipMemcpyHostToDevice);
	calculate<<<1, nThreads>>> (argsH);
	hipMemcpy(args, argsH, nThreads * sizeof(CalcArgs), hipMemcpyDeviceToHost);
	
	hipDeviceSynchronize();

	for (int i = 0; i < nThreads; i++)
	{
		//_(pthread_join(args[i].tid, NULL));
		sum += args[i].sum;
	}

	printf("%lg\n", sum);

	return 0;
}
